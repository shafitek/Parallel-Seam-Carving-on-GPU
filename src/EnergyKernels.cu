#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <>

#include "EnergyKernels.cuh"

#define BLK_WIDTH 32
#define BLK_HEIGHT 32
#define BLK_CHANNEL 3

typedef unsigned char uint_8;

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void calcEnergyImg(double *img, double *energy_img);
__global__ void calcEnergyMap(double *img, double *energy_map);

__global__ void convertBGR2GRAYGPU(uint_8 *img, const int img_step, uint_8 *res, const int res_step, int width)
{
    int c_col = blockIdx.x * blockDim.x + threadIdx.x;

    if(c_col >= width) return;

    int c_row = blockIdx.y * blockDim.y + threadIdx.y;
    int gid = img_step * c_row + c_col * 3;
    int gid_res = res_step * c_row + c_col;
    
    double px = (img[gid] + img[gid+1] + img[gid+2]) / 3.0;
    res[gid_res] = (uint_8) px;
}

__global__ void computeEnergyImageGPU(uint_8 *img, const int img_step, uint_8 *res, const int res_step, int width, int height)
{
    int c_col = blockIdx.x * blockDim.x + threadIdx.x;

    if(c_col >= width) return;

    int c_row = blockIdx.y * blockDim.y + threadIdx.y;
    int gid = img_step * c_row + c_col;
    int gid_r = res_step * c_row + c_col;

    uint_8 xg = c_col != width;
    uint_8 yg = c_row != height;

    int diffx, diffy, diffxy, tmpPx;
    diffx = xg ? abs(img[gid] - img[gid+1]) : 0;
    diffy = yg ? abs(img[gid] - img[gid + img_step]) : 0;
    diffxy = xg ? abs(img[gid] - img[gid + img_step + 1]) : 0;
    tmpPx = min(255, diffx + diffy + diffxy);

    res[gid_r] = (uint_8)tmpPx;
}

__global__ void copyFirstRow(uint_8 *img, double *res, int width)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if(gid >= width) return;

    res[gid] = img[gid];
}

__global__ void computeEnergyMapGPU(uint_8 *img, const int img_step, double *res, const int res_step, int width, int row, int block_num)
{
    unsigned int col = block_num * blockDim.x + threadIdx.x;

    if(col >= width) return;

    int gid = row * img_step + col;

    // https://stackoverflow.com/questions/44174092/how-to-multiply-two-opencv-matrices-in-a-kernel-function-in-cuda
    // Access the double array like ((double *)((unsigned char *)res + row * res_step))[col].

    double left, middle, right;
    left = col % width != 0 ? ((double *)((unsigned char *)res + (row - 1) * res_step))[col - 1] : INT_MAX;
    middle = ((double *)((unsigned char *)res + (row - 1) * res_step))[col];
    right = (width - 1) != col ? ((double *)((unsigned char *)res + (row - 1) * res_step))[col + 1] : INT_MAX;

    ((double *)((unsigned char *)res + row * res_step))[col] 
            = img[gid] + min(middle, min(left, right));
}

void computeEnergyMapGPU(const cv::Mat &c_imgi, cv::Mat &f_ener_img, cv::Mat &f_ener_map, double &time)
{
    uint8_t *img_ptr, *gry_ptr;
    cv::Mat c_img = c_imgi;
    f_ener_img.create(c_img.rows, c_img.cols, CV_8U);
    f_ener_map.create(c_img.rows, c_img.cols, CV_64F);
    cv::Mat &c_ener_img = f_ener_img;
    cv::Mat &c_ener_map = f_ener_map;

    auto start = std::chrono::high_resolution_clock::now();
    cv::cuda::GpuMat g_img;
    g_img.upload(c_img);

    dim3 block(BLK_WIDTH, BLK_HEIGHT);
    dim3 grid((g_img.cols + BLK_WIDTH - 1) / BLK_WIDTH, (c_img.rows + BLK_WIDTH - 1) / BLK_HEIGHT);

    hipMalloc((void **)&img_ptr, g_img.rows * g_img.step);
    hipMemcpyAsync(img_ptr, g_img.ptr<uint8_t>(), g_img.rows * g_img.step, hipMemcpyDeviceToDevice);

    // cv::cuda::GpuMat g_gry_img(c_img.rows, c_img.cols, c_ener_img.type(), img_ptr, c_ener_img.step);

    // convertBGR2GRAYGPU<<<grid, block>>>(
    //     (uint_8 *)g_img.data, g_img.step,
    //     (uint_8 *)g_gry_img.data, g_gry_img.step,
    //     c_img.cols);

    // gry_ptr = g_gry_img.ptr<uint8_t>();
    cv::cuda::GpuMat g_ener_img(c_img.rows, c_img.cols, c_ener_img.type(), img_ptr, c_ener_img.step);

    computeEnergyImageGPU<<<grid, block>>>(
        (uint_8 *)g_img.data, g_img.step,
        (uint_8 *)g_ener_img.data, g_ener_img.step,
        c_img.cols, c_img.rows);

    cv::cuda::GpuMat g_ener_map(c_img.rows, c_img.cols, CV_64F);

    copyFirstRow<<<(c_img.cols  + BLK_WIDTH - 1) / BLK_WIDTH, BLK_WIDTH>>>(
        (uint_8 *)g_ener_img.data,
        (double *)g_ener_map.data,
        c_img.cols);

    hipDeviceSynchronize();

    const size_t num_streams = (c_img.cols + BLK_WIDTH - 1) / BLK_WIDTH;

    for(int row = 1; row < c_img.rows; row++) {
        hipStream_t streams[num_streams];
        for (int st = 0; st < num_streams; st++)
        {
            hipStreamCreate(&streams[st]);
            computeEnergyMapGPU<<<1, BLK_WIDTH, 0, streams[st]>>>(
                (uint_8 *)g_ener_img.data, g_ener_img.step,
                (double *)g_ener_map.data, g_ener_map.step,
                c_img.cols, row, st);
        }
        for (int sd = 0; sd < num_streams; sd++)
            hipStreamDestroy(streams[sd]);
    }

    g_ener_img.download(c_ener_img);
    g_ener_map.download(c_ener_map);
    // gpuErrchk(hipPeekAtLastError());

    auto end = std::chrono::high_resolution_clock::now();

    time = (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    hipFree(img_ptr);
    hipFree(gry_ptr);

    hipDeviceReset();
}